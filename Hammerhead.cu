#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sys/time.h"
#include "string.h"
#include <stdbool.h>
#include <math.h>

#include <thrust/host_vector.h>

// All Constants, Globals, Enums

// Used for low level chess functions, such as adding a move, determining if a piece is under attack, etc.
// Some functions are used on the device, so they have a device version

enum side
{
    w,
    b
};
enum pieces
{
    emSq,
    wP,
    wN,
    wB,
    wR,
    wQ,
    wK,
    offBoard = 8,
    bP,
    bN,
    bB,
    bR,
    bQ,
    bK
};
enum castling
{
    K = 1,
    Q = 2,
    k = 4,
    q = 8
};

__device__ enum dev_castling { dev_K = 1,
                               dev_Q = 2,
                               dev_k = 4,
                               dev_q = 8 };

enum squares
{
    a1 = 0,
    b1,
    c1,
    d1,
    e1,
    f1,
    g1,
    h1,
    a2 = 16,
    b2,
    c2,
    d2,
    e2,
    f2,
    g2,
    h2,
    a3 = 32,
    b3,
    c3,
    d3,
    e3,
    f3,
    g3,
    h3,
    a4 = 48,
    b4,
    c4,
    d4,
    e4,
    f4,
    g4,
    h4,
    a5 = 64,
    b5,
    c5,
    d5,
    e5,
    f5,
    g5,
    h5,
    a6 = 80,
    b6,
    c6,
    d6,
    e6,
    f6,
    g6,
    h6,
    a7 = 96,
    b7,
    c7,
    d7,
    e7,
    f7,
    g7,
    h7,
    a8 = 112,
    b8,
    c8,
    d8,
    e8,
    f8,
    g8,
    h8,
    noSq = -99
};

enum moveFlags
{
    allPos,
    captures
};

// attack directions
const int pawnAttacks[4] = {15, 17, -15, -17};
const int knightAttacks[8] = {31, 33, 14, 18, -31, -33, -14, -18};
const int kingAttacks[8] = {1, 15, 16, 17, -1, -15, -16, -17};
const int bishopAttacks[4] = {15, 17, -15, -17};
const int rookAttacks[4] = {1, 16, -1, -16};

__device__ const int dev_pawnAttacks[4] = {15, 17, -15, -17};
__device__ const int dev_knightAttacks[8] = {31, 33, 14, 18, -31, -33, -14, -18};
__device__ const int dev_kingAttacks[8] = {1, 15, 16, 17, -1, -15, -16, -17};
__device__ const int dev_bishopAttacks[4] = {15, 17, -15, -17};
__device__ const int dev_rookAttacks[4] = {1, 16, -1, -16};

// tracking whether kings or rooks moved
const int castling[128] =
    {
        13, 15, 15, 15, 12, 15, 15, 14, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        7, 15, 15, 15, 3, 15, 15, 11, 8, 8, 8, 8, 8, 8, 8, 8};

__device__ const int dev_castling[128] =
    {
        13, 15, 15, 15, 12, 15, 15, 14, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        15, 15, 15, 15, 15, 15, 15, 15, 8, 8, 8, 8, 8, 8, 8, 8,
        7, 15, 15, 15, 3, 15, 15, 11, 8, 8, 8, 8, 8, 8, 8, 8};

// material weight of pieces
const int materialWeight[15] =
    {
        0, 100, 300, 350, 525, 1000, 10000, 0,
        0, -100, -300, -350, -525, -1000, -10000};

__device__ const int dev_materialWeight[15] =
    {
        0, 100, 300, 350, 525, 1000, 10000, 0,
        0, -100, -300, -350, -525, -1000, -10000};

// piece placement tables, as pieces have different values depending on their position
const int Pawns[128] =
    {
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, -10, -10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        5, 5, 5, 5, 5, 5, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        5, 5, 5, 20, 20, 5, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        10, 10, 10, 20, 20, 10, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0,
        10, 10, 10, 20, 20, 10, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0,
        20, 20, 20, 30, 30, 20, 20, 20, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

const int Knights[128] =
    {
        0, -10, 0, 0, 0, 0, -10, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 10, 10, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 10, 20, 20, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        5, 10, 15, 20, 20, 15, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        5, 10, 10, 20, 20, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

const int Bishops[128] =
    {
        0, 0, -10, 0, 0, -10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 10, 15, 15, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 10, 20, 20, 20, 20, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 10, 15, 20, 20, 15, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 10, 15, 15, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

const int Rooks[128] =
    {
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        25, 25, 25, 25, 25, 25, 25, 25, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

const int Kings[128] =
    {
        5, 5, 0, -10, -10, 0, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 5, -10, -10, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 10, 10, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 20, 20, 20, 20, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 20, 20, 20, 20, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 10, 10, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 5, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

// Mirror evaluation tables for opposite side
const int Mirror[128] =
    {
        a8, b8, c8, d8, e8, f8, g8, h8, 0, 0, 0, 0, 0, 0, 0, 0,
        a7, b7, c7, d7, e7, f7, g7, h7, 0, 0, 0, 0, 0, 0, 0, 0,
        a6, b6, c6, d6, e6, f6, g6, h6, 0, 0, 0, 0, 0, 0, 0, 0,
        a5, b5, c5, d5, e5, f5, g5, h5, 0, 0, 0, 0, 0, 0, 0, 0,
        a4, b4, c4, d4, e4, f4, g4, h4, 0, 0, 0, 0, 0, 0, 0, 0,
        a3, b3, c3, d3, e3, f3, g3, h3, 0, 0, 0, 0, 0, 0, 0, 0,
        a2, b2, c2, d2, e2, f2, g2, h2, 0, 0, 0, 0, 0, 0, 0, 0,
        a1, b1, c1, d1, e1, f1, g1, h1, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ const int dev_Pawns[128] =
    {
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, -10, -10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        5, 5, 5, 5, 5, 5, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        5, 5, 5, 20, 20, 5, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        10, 10, 10, 20, 20, 10, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0,
        10, 10, 10, 20, 20, 10, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0,
        20, 20, 20, 30, 30, 20, 20, 20, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ const int dev_Knights[128] =
    {
        0, -10, 0, 0, 0, 0, -10, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 10, 10, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 10, 20, 20, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        5, 10, 15, 20, 20, 15, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        5, 10, 10, 20, 20, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ const int dev_Bishops[128] =
    {
        0, 0, -10, 0, 0, -10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 10, 15, 15, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 10, 20, 20, 20, 20, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 10, 15, 20, 20, 15, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 10, 15, 15, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 10, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ const int dev_Rooks[128] =
    {
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        25, 25, 25, 25, 25, 25, 25, 25, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ const int dev_Kings[128] =
    {
        5, 5, 0, -10, -10, 0, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 5, -10, -10, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 10, 10, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 20, 20, 20, 20, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 20, 20, 20, 20, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 5, 10, 10, 10, 10, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 5, 5, 5, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ const int dev_Mirror[128] =
    {
        a8, b8, c8, d8, e8, f8, g8, h8, 0, 0, 0, 0, 0, 0, 0, 0,
        a7, b7, c7, d7, e7, f7, g7, h7, 0, 0, 0, 0, 0, 0, 0, 0,
        a6, b6, c6, d6, e6, f6, g6, h6, 0, 0, 0, 0, 0, 0, 0, 0,
        a5, b5, c5, d5, e5, f5, g5, h5, 0, 0, 0, 0, 0, 0, 0, 0,
        a4, b4, c4, d4, e4, f4, g4, h4, 0, 0, 0, 0, 0, 0, 0, 0,
        a3, b3, c3, d3, e3, f3, g3, h3, 0, 0, 0, 0, 0, 0, 0, 0,
        a2, b2, c2, d2, e2, f2, g2, h2, 0, 0, 0, 0, 0, 0, 0, 0,
        a1, b1, c1, d1, e1, f1, g1, h1, 0, 0, 0, 0, 0, 0, 0, 0};

// Structs used for represending the board, search, and possible moves
typedef struct
{
    int move;
    int score;
} Move;
typedef struct
{
    Move moves[256];
    int moveCount;
} Movelist;

typedef struct
{
    int position[128];

    int side;
    int enPassant;
    int castle;

    int kingSq[2];

    int ply;
}

Chessboard;

typedef struct
{
    long nodes;
    double fhf, fh;

    int bestMove;
    int bestScore;

}

Search;

// Global vectors that hold all the boards and searches that get sent to the GPU
// They are global as I was having bugs with passing them by reference and security or missuse is not an issue here.

thrust::host_vector<Chessboard> boards;
thrust::host_vector<Search> searches;

// The following are a series of Macros that significantly clean up code presentation
// I may have gotten carried away ...

#define MirrorSq(sq) Mirror[sq]
#define DevMirrorSq(sq) dev_Mirror[sq]

// 0x88 math
#define IsOnBoard(sq) (!(sq & 0x88))
#define fr2sq(file, rank) (rank * 16 - file)
#define parse2sq(file, rank) ((rank - 1) * 16 + file)
#define GetFile(sq) (sq & 7)
#define GetRank(sq) (sq >> 4)
#define rank_7 (fromSq >= a7 && fromSq <= h7)
#define rank_2 (fromSq >= a2 && fromSq <= h2)

// Convertions
#define GetFileChar(sq) (GetFile(sq) + 'a')
#define GetRankChar(sq) (GetRank(sq) + '1')

// Char type
#define isDigit(char) (char >= '0' && char <= '9')
#define isPieceChar(piece) ((*fen >= 'a' && *fen <= 'z') || ((*fen >= 'A' && *fen <= 'Z')))

// White or black
#define isBlack(toSq) (board->position[toSq] >= bN && board->position[toSq] <= bQ)
#define isWhite(toSq) (board->position[toSq] >= wN && board->position[toSq] <= wQ)

#define isBlackPiece(toSq) (board->position[toSq] >= bP && board->position[toSq] <= bK)
#define DevisBlackPiece(toSq) (board->position[toSq] >= bP && board->position[toSq] <= bK)

#define isWhitePiece(toSq) (board->position[toSq] >= wP && board->position[toSq] <= wK)
#define DevisWhitePiece(toSq) (board->position[toSq] >= wP && board->position[toSq] <= wK)

// Quick board access
#define pos(sq) board->position[sq]
#define side board->side
#define enPassant board->enPassant
#define castle board->castle
#define kingSq(col) board->kingSq[col]
#define DevkingSq(col) board->kingSq[col]

#define ply board->ply

// Board loops
#define LoopBoard for (int sq = 0; sq < 128; ++sq)
#define RankLoop for (int rank = 8; rank >= 1; rank--)
#define FileLoop for (int file = 16; file >= 1; file--)

// Board methods
#define SetSq(sq, piece) (pos(sq) = piece)
#define DevSetSq(sq, piece) (pos(sq) = piece)

#define GetSq(sq) pos(sq)
#define DevGetSq(sq) pos(sq)

#define PrintSquare(sq) \
    if (sq == -99)      \
        printf("no");   \
    else                \
        printf("%c%c", GetFileChar(sq), GetRankChar(sq));
// #define PrintPromotedPiece(piece) printf("%c", promotedPieceChar[piece])

// Init board
#define ResetPosition(board) \
    LoopBoard { IsOnBoard(sq) ? SetSq(sq, emSq) : SetSq(sq, offBoard); }

#define ResetStats(board) \
    side = 0;             \
    enPassant = noSq;     \
    castle = 0;           \
    ply = 0;

#define ResetBoard(board) \
    ResetPosition(board); \
    ResetStats(board)

// Print board
#define PrintPosition(board)                              \
    printf("\n");                                         \
    RankLoop                                              \
    {                                                     \
        printf("  %d", rank);                             \
        FileLoop                                          \
        {                                                 \
            if (GetSq(fr2sq(file, rank)) != 8)            \
                printf("  %i", GetSq(fr2sq(file, rank))); \
        }                                                 \
        printf("\n");                                     \
    }

#define PrintStats(board)                                           \
    printf("\n     a  b  c  d  e  f  g  h\n\n");                    \
    printf("     Side:            %s\n", side ? "black" : "white"); \
    printf("     EnPassant:          ");                            \
    PrintSquare(enPassant);                                         \
    printf("\n");                                                   \
    printf("     Castling:         %c%c%c%c\n",                     \
           castle &K ? 'K' : '-',                                   \
           castle &Q ? 'Q' : '-',                                   \
           castle &k ? 'k' : '-',                                   \
           castle &q ? 'q' : '-');                                  \
    printf("\n\n");

#define PrintBoard(board) \
    PrintPosition(board); \
    PrintStats(board);

#define DevPrintPosition(board)                           \
    printf("\n");                                         \
    RankLoop                                              \
    {                                                     \
        printf("  %d", rank);                             \
        FileLoop                                          \
        {                                                 \
            if (GetSq(fr2sq(file, rank)) != 8)            \
                printf("  %i", GetSq(fr2sq(file, rank))); \
        }                                                 \
        printf("\n");                                     \
    }

#define DevPrintStats(dev_board)                                    \
    printf("\n     a  b  c  d  e  f  g  h\n\n");                    \
    printf("     Side:            %s\n", side ? "black" : "white"); \
    printf("     EnPassant:          ");                            \
    PrintSquare(enPassant);                                         \
    printf("\n");                                                   \
    printf("     Castling:         %c%c%c%c\n",                     \
           castle &K ? 'K' : '-',                                   \
           castle &Q ? 'Q' : '-',                                   \
           castle &k ? 'k' : '-',                                   \
           castle &q ? 'q' : '-');                                  \
    printf("\n\n");

#define DevPrintBoard(dev_board) \
    DevPrintPosition(dev_board); \
    DevPrintStats(dev_board);

#define SetMove(f, t, prom, cap, pawn, e, cas) \
    ((f) | (t << 7) | (prom << 14) | (cap << 18) | (pawn << 19) | (e << 20) | (cas << 21))
#define DevSetMove(f, t, prom, cap, pawn, e, cas) \
    ((f) | (t << 7) | (prom << 14) | (cap << 18) | (pawn << 19) | (e << 20) | (cas << 21))

#define GetMoveSource(move) (move & 0x7f)
#define DevGetMoveSource(move) (move & 0x7f)

#define GetMoveTarget(move) ((move >> 7) & 0x7f)
#define DevGetMoveTarget(move) ((move >> 7) & 0x7f)

#define GetMovePromPiece(move) ((move >> 14) & 0xf)
#define DevGetMovePromPiece(move) ((move >> 14) & 0xf)

#define GetMoveCaptureFlag(move) ((move >> 18) & 1)
#define DevGetMoveCaptureFlag(move) ((move >> 18) & 1)

#define GetMovePawnStartFlag(move) ((move >> 19) & 1)
#define DevGetMovePawnStartFlag(move) ((move >> 19) & 1)

#define GetMoveEnPassantFlag(move) ((move >> 20) & 1)
#define DevGetMoveEnPassantFlag(move) ((move >> 20) & 1)

#define GetMoveCastleFlag(move) ((move >> 21) & 1)
#define DevGetMoveCastleFlag(move) ((move >> 21) & 1)

#define SortMoves                                                            \
    for (int nextMove = moveNum + 1; nextMove < list->moveCount; ++nextMove) \
    {                                                                        \
        if (list->moves[moveNum].score < list->moves[nextMove].score)        \
        {                                                                    \
            int tempScore = list->moves[moveNum].score;                      \
            int tempMove = list->moves[moveNum].move;                        \
            list->moves[moveNum].score = list->moves[nextMove].score;        \
            list->moves[nextMove].score = tempScore;                         \
            list->moves[moveNum].move = list->moves[nextMove].move;          \
            list->moves[nextMove].move = tempMove;                           \
        }                                                                    \
    }

#define PrintMove(move)               \
    printf(" ");                      \
    PrintSquare(GetMoveSource(move)); \
    PrintSquare(GetMoveTarget(move));

#define LoopMoves for (int moveCount = 0; moveCount < list->moveCount; ++moveCount)

#define PrintMoveList(list)                                      \
    LoopMoves                                                    \
    {                                                            \
        PrintMove(list->moves[moveCount].move);                  \
        printf("	SCORE: %d\n", list->moves[moveCount].score); \
    }                                                            \
    printf("\n  Total moves: %d\n\n", list->moveCount);

#define TakeBack(board, boardStored) board[0] = boardStored[0];
#define DevTakeBack(board, boardStored) board[0] = boardStored[0];

#define InCheck(board, sideToMove) \
    IsSquareAttacked(board, sideToMove ? kingSq(b) : kingSq(w), sideToMove ^ 1)

#define DevInCheck(board, sideToMove) \
    DevIsSquareAttacked(board, sideToMove ? kingSq(b) : kingSq(w), sideToMove ^ 1)

/*****The real fun begins here!*****/

// Function that determines whether a piece on a given square is being attacked
// The function works by checking all possible positions that a given piece type can attack the given piece.
static inline int IsSquareAttacked(Chessboard *board, int sq, int attSide)
{
    // by pawns
    if (!attSide)
    {
        if (!((sq - 15) & 0x88) && (GetSq(sq - 15) == wP))
            return 1;

        if (!((sq - 17) & 0x88) && (GetSq(sq - 17) == wP))
            return 1;
    }

    else
    {
        if (!((sq + 15) & 0x88) && (GetSq(sq + 15) == bP))
            return 1;

        if (!((sq + 17) & 0x88) && (GetSq(sq + 17) == bP))
            return 1;
    }

    // by knights
    for (int i = 0; i < 8; ++i)
    {
        int dir = sq + knightAttacks[i];
        int delta = GetSq(dir);

        if (!(dir & 0x88))
        {
            if (attSide ? delta == bN : delta == wN)
                return 1;
        }
    }

    // by bishops and queens
    for (int i = 0; i < 4; ++i)
    {
        int dir = sq + bishopAttacks[i];

        while (!(dir & 0x88))
        {
            int delta = GetSq(dir);

            if (attSide ? (delta == bB) || (delta == bQ) : (delta == wB) || (delta == wQ))
                return 1;

            else if (delta != 0)
                break;

            dir += bishopAttacks[i];
        }
    }

    // by rooks and queens
    for (int i = 0; i < 4; ++i)
    {
        int dir = sq + rookAttacks[i];

        while (!(dir & 0x88))
        {
            int delta = GetSq(dir);

            if (attSide ? (delta == bR) || (delta == bQ) : (delta == wR) || (delta == wQ))
                return 1;

            else if (delta != 0)
                break;

            dir += rookAttacks[i];
        }
    }

    // by kings
    for (int i = 0; i < 8; ++i)
    {
        int dir = sq + kingAttacks[i];
        int delta = GetSq(dir);

        if (!(dir & 0x88))
        {
            if (attSide ? delta == bK : delta == wK)
                return 1;
        }
    }

    return 0;
}

// device version of the function above
__device__ static inline int DevIsSquareAttacked(Chessboard *board, int sq, int attSide)
{
    // by pawns
    if (!attSide)
    {
        if (!((sq - 15) & 0x88) && (GetSq(sq - 15) == wP))
            return 1;

        if (!((sq - 17) & 0x88) && (GetSq(sq - 17) == wP))
            return 1;
    }

    else
    {
        if (!((sq + 15) & 0x88) && (GetSq(sq + 15) == bP))
            return 1;

        if (!((sq + 17) & 0x88) && (GetSq(sq + 17) == bP))
            return 1;
    }

    // by knights
    for (int i = 0; i < 8; ++i)
    {
        int dir = sq + dev_knightAttacks[i];
        int delta = GetSq(dir);

        if (!(dir & 0x88))
        {
            if (attSide ? delta == bN : delta == wN)
                return 1;
        }
    }

    // by bishops and queens
    for (int i = 0; i < 4; ++i)
    {
        int dir = sq + dev_bishopAttacks[i];

        while (!(dir & 0x88))
        {
            int delta = GetSq(dir);

            if (attSide ? (delta == bB) || (delta == bQ) : (delta == wB) || (delta == wQ))
                return 1;

            else if (delta != 0)
                break;

            dir += dev_bishopAttacks[i];
        }
    }

    // by rooks and queens
    for (int i = 0; i < 4; ++i)
    {
        int dir = sq + dev_rookAttacks[i];

        while (!(dir & 0x88))
        {
            int delta = GetSq(dir);

            if (attSide ? (delta == bR) || (delta == bQ) : (delta == wR) || (delta == wQ))
                return 1;

            else if (delta != 0)
                break;

            dir += dev_rookAttacks[i];
        }
    }

    // by kings
    for (int i = 0; i < 8; ++i)
    {
        int dir = sq + dev_kingAttacks[i];
        int delta = GetSq(dir);

        if (!(dir & 0x88))
        {
            if (attSide ? delta == bK : delta == wK)
                return 1;
        }
    }

    return 0;
}

// Function that adds a move to our list of moves.
static inline void AddMove(Chessboard *board, Search *info, Movelist *list, int move)
{
    list->moves[list->moveCount].move = move;

    list->moveCount++;
}

// device version of the function above
__device__ static inline void DevAddMove(Chessboard *board, Search *info, Movelist *list, int move)
{
    //printf("here");

    list->moves[list->moveCount].move = move;

    list->moveCount++;
}

// Generates all possible moves, some of which are illegal
// The function works by looping through all squares and on each square checking
// the type of piece it is and then from there it adds all moves to our search and list
static inline void GenerateMoves(Chessboard *board, Search *info, Movelist *list)
{
    list->moveCount = 0;

    for (int sq = 0; sq < 128; ++sq)
    {
        if (!(sq & 0x88))
        {
            // skip empty squares
            if (!GetSq(sq))
                continue;

            int fromSq = sq;

            if (!side)
            {
                if (GetSq(fromSq) == wP)
                {
                    // pawn quiet move
                    if (!((fromSq + 16) & 0x88) && !GetSq(fromSq + 16))
                    {
                        if (rank_7 && !GetSq(fromSq + 16))
                        {
                            AddMove(board, info, list, SetMove(fromSq, fromSq + 16, wN, 0, 0, 0, 0));
                            AddMove(board, info, list, SetMove(fromSq, fromSq + 16, wB, 0, 0, 0, 0));
                            AddMove(board, info, list, SetMove(fromSq, fromSq + 16, wR, 0, 0, 0, 0));
                            AddMove(board, info, list, SetMove(fromSq, fromSq + 16, wQ, 0, 0, 0, 0));
                        }

                        else
                        {
                            AddMove(board, info, list, SetMove(fromSq, fromSq + 16, 0, 0, 0, 0, 0));

                            if (rank_2 && !GetSq(fromSq + 32))
                                AddMove(board, info, list, SetMove(fromSq, fromSq + 32, 0, 0, 1, 0, 0));
                        }
                    }

                    // pawn capture move
                    for (int i = 0; i < 4; ++i)
                    {
                        int dir = fromSq + pawnAttacks[i];

                        // en passant move
                        if (pawnAttacks[i] > 0 && !(dir & 0x88))
                        {
                            if (enPassant != noSq)
                            {
                                if (dir == enPassant)
                                    AddMove(board, info, list, SetMove(fromSq, dir, 0, 1, 0, 1, 0));
                            }
                        }

                        if ((pawnAttacks[i] > 0) && !(dir & 0x88) && isBlackPiece(dir))
                        {
                            if (rank_7)
                            {
                                AddMove(board, info, list, SetMove(fromSq, dir, wN, 1, 0, 0, 0));
                                AddMove(board, info, list, SetMove(fromSq, dir, wB, 1, 0, 0, 0));
                                AddMove(board, info, list, SetMove(fromSq, dir, wR, 1, 0, 0, 0));
                                AddMove(board, info, list, SetMove(fromSq, dir, wQ, 1, 0, 0, 0));
                            }

                            else
                            {
                                AddMove(board, info, list, SetMove(fromSq, dir, 0, 1, 0, 0, 0));
                            }
                        }
                    }
                }

                // castling
                if (GetSq(fromSq) == wK)
                {
                    if (castle & K)
                    {
                        if (!GetSq(f1) && !GetSq(g1))
                        {
                            if (!IsSquareAttacked(board, e1, b) && !IsSquareAttacked(board, f1, b))
                                AddMove(board, info, list, SetMove(e1, g1, 0, 0, 0, 0, 1));
                        }
                    }

                    if (castle & Q)
                    {
                        if (!GetSq(d1) && !GetSq(c1) && !GetSq(b1))
                        {
                            if (!IsSquareAttacked(board, e1, b) && !IsSquareAttacked(board, d1, b))
                                AddMove(board, info, list, SetMove(e1, c1, 0, 0, 0, 0, 1));
                        }
                    }
                }
            }

            else
            {
                if (GetSq(fromSq) == bP)
                {
                    // pawn quiet move
                    if (!((fromSq - 16) & 0x88) && !GetSq(fromSq - 16))
                    {
                        if (rank_2 && !GetSq(fromSq - 16))
                        {
                            AddMove(board, info, list, SetMove(fromSq, fromSq - 16, bN, 0, 0, 0, 0));
                            AddMove(board, info, list, SetMove(fromSq, fromSq - 16, bB, 0, 0, 0, 0));
                            AddMove(board, info, list, SetMove(fromSq, fromSq - 16, bR, 0, 0, 0, 0));
                            AddMove(board, info, list, SetMove(fromSq, fromSq - 16, bQ, 0, 0, 0, 0));
                        }

                        else
                        {
                            AddMove(board, info, list, SetMove(fromSq, fromSq - 16, 0, 0, 0, 0, 0));

                            if (rank_7 && !GetSq(fromSq - 32))
                                AddMove(board, info, list, SetMove(fromSq, fromSq - 32, 0, 0, 1, 0, 0));
                        }
                    }

                    // pawn capture move
                    for (int i = 0; i < 4; ++i)
                    {
                        int dir = fromSq + pawnAttacks[i];

                        // en passant move
                        if (pawnAttacks[i] < 0 && !(dir & 0x88))
                        {
                            if (enPassant != noSq)
                            {
                                if (dir == enPassant)
                                    AddMove(board, info, list, SetMove(fromSq, dir, 0, 0, 0, 1, 0));
                            }
                        }

                        if ((pawnAttacks[i] < 0) && !(dir & 0x88) && isWhitePiece(dir))
                        {
                            if (rank_2)
                            {
                                AddMove(board, info, list, SetMove(fromSq, dir, bN, 1, 0, 0, 0));
                                AddMove(board, info, list, SetMove(fromSq, dir, bB, 1, 0, 0, 0));
                                AddMove(board, info, list, SetMove(fromSq, dir, bR, 1, 0, 0, 0));
                                AddMove(board, info, list, SetMove(fromSq, dir, bQ, 1, 0, 0, 0));
                            }

                            else
                            {
                                AddMove(board, info, list, SetMove(fromSq, dir, 0, 1, 0, 0, 0));
                            }
                        }
                    }
                }

                // castling
                if (GetSq(fromSq) == bK)
                {
                    if (castle & k)
                    {
                        if (!GetSq(f8) && !GetSq(g8))
                        {
                            if (!IsSquareAttacked(board, e8, w) && !IsSquareAttacked(board, f8, w))
                                AddMove(board, info, list, SetMove(e8, g8, 0, 0, 0, 0, 1));
                        }
                    }

                    if (castle & q)
                    {
                        if (!GetSq(d8) && !GetSq(c8) && !GetSq(b8))
                        {
                            if (!IsSquareAttacked(board, e8, w) && !IsSquareAttacked(board, d8, w))
                                AddMove(board, info, list, SetMove(e8, c8, 0, 0, 0, 0, 1));
                        }
                    }
                }
            }

            // knights
            if (side ? GetSq(fromSq) == bN : GetSq(fromSq) == wN)
            {
                for (int i = 0; i < 8; ++i)
                {
                    int dir = sq + knightAttacks[i];
                    int delta = GetSq(dir);

                    if (!(dir & 0x88))
                    {
                        if (side ? (!delta || isWhitePiece(dir)) : (!delta || isBlackPiece(dir)))
                        {
                            if (!delta)
                                AddMove(board, info, list, SetMove(fromSq, dir, 0, 0, 0, 0, 0));
                            else
                                AddMove(board, info, list, SetMove(fromSq, dir, 0, 1, 0, 0, 0));
                        }
                    }
                }
            }

            // bishops and queens
            if (side ? (GetSq(fromSq) == bB) || (GetSq(fromSq) == bQ) : (GetSq(fromSq) == wB) || (GetSq(fromSq) == wQ))

            {
                for (int i = 0; i < 4; ++i)
                {
                    int dir = sq + bishopAttacks[i];

                    while (!(dir & 0x88))
                    {
                        int delta = GetSq(dir);

                        // if hits own piece
                        if (side ? isBlackPiece(dir) : isWhitePiece(dir))
                            break;

                        // if hits opponent's piece
                        else if (side ? isWhitePiece(dir) : isBlackPiece(dir))
                        {
                            AddMove(board, info, list, SetMove(fromSq, dir, 0, 1, 0, 0, 0));
                            break;
                        }

                        // on empty square
                        else if (!delta)
                        {
                            AddMove(board, info, list, SetMove(fromSq, dir, 0, 0, 0, 0, 0));
                        }

                        dir += bishopAttacks[i];
                    }
                }
            }

            // rooks and queens
            if (side ? (GetSq(fromSq) == bR) || (GetSq(fromSq) == bQ) : (GetSq(fromSq) == wR) || (GetSq(fromSq) == wQ))

            {
                for (int i = 0; i < 4; ++i)
                {
                    int dir = sq + rookAttacks[i];

                    while (!(dir & 0x88))
                    {
                        int delta = GetSq(dir);

                        // if hits own piece
                        if (side ? isBlackPiece(dir) : isWhitePiece(dir))
                            break;

                        // if hits opponent's piece
                        else if (side ? isWhitePiece(dir) : isBlackPiece(dir))
                        {
                            AddMove(board, info, list, SetMove(fromSq, dir, 0, 1, 0, 0, 0));
                            break;
                        }

                        // on empty square
                        else if (!delta)
                        {
                            AddMove(board, info, list, SetMove(fromSq, dir, 0, 0, 0, 0, 0));
                        }

                        dir += rookAttacks[i];
                    }
                }
            }

            // kings
            if (side ? GetSq(fromSq) == bK : GetSq(fromSq) == wK)
            {
                for (int i = 0; i < 8; ++i)
                {
                    int dir = sq + kingAttacks[i];
                    int delta = GetSq(dir);

                    if (!(dir & 0x88))
                    {
                        if (side ? (!delta || isWhitePiece(dir)) : (!delta || isBlackPiece(dir)))
                        {
                            if (!delta)
                                AddMove(board, info, list, SetMove(fromSq, dir, 0, 0, 0, 0, 0));
                            else
                                AddMove(board, info, list, SetMove(fromSq, dir, 0, 1, 0, 0, 0));
                        }
                    }
                }
            }
        }
    }
}

// device version of the function above
__device__ static void DevGenerateMoves(Chessboard *board, Search *info, Movelist *list)
{
    list->moveCount = 0;
    //printf("some\n");

    for (int sq = 0; sq < 128; ++sq)
    {
        if (!(sq & 0x88))
        {
            // skip empty squares
            if (!DevGetSq(sq))
                continue;

            int fromSq = sq;

            if (!side)
            {
                if (DevGetSq(fromSq) == wP)
                {
                    // pawn quiet move
                    if (!((fromSq + 16) & 0x88) && !DevGetSq(fromSq + 16))
                    {
                        if (rank_7 && !DevGetSq(fromSq + 16))
                        {
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq + 16, wN, 0, 0, 0, 0));
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq + 16, wB, 0, 0, 0, 0));
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq + 16, wR, 0, 0, 0, 0));
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq + 16, wQ, 0, 0, 0, 0));
                        }

                        else
                        {
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq + 16, 0, 0, 0, 0, 0));

                            if (rank_2 && !DevGetSq(fromSq + 32))
                                DevAddMove(board, info, list, DevSetMove(fromSq, fromSq + 32, 0, 0, 1, 0, 0));
                        }
                    }

                    // pawn capture move
                    for (int i = 0; i < 4; ++i)
                    {
                        int dir = fromSq + dev_pawnAttacks[i];

                        // en passant move
                        if (dev_pawnAttacks[i] > 0 && !(dir & 0x88))
                        {
                            if (enPassant != noSq)
                            {
                                if (dir == enPassant)
                                    DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 1, 0, 1, 0));
                            }
                        }

                        if ((dev_pawnAttacks[i] > 0) && !(dir & 0x88) && isBlackPiece(dir))
                        {
                            if (rank_7)
                            {
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, wN, 1, 0, 0, 0));
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, wB, 1, 0, 0, 0));
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, wR, 1, 0, 0, 0));
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, wQ, 1, 0, 0, 0));
                            }

                            else
                            {
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 1, 0, 0, 0));
                            }
                        }
                    }
                }

                // castling
                if (DevGetSq(fromSq) == wK)
                {
                    if (castle & K)
                    {
                        if (!DevGetSq(f1) && !DevGetSq(g1))
                        {
                            if (!DevIsSquareAttacked(board, e1, b) && !DevIsSquareAttacked(board, f1, b))
                                DevAddMove(board, info, list, DevSetMove(e1, g1, 0, 0, 0, 0, 1));
                        }
                    }

                    if (castle & Q)
                    {
                        if (!DevGetSq(d1) && !DevGetSq(c1) && !DevGetSq(b1))
                        {
                            if (!DevIsSquareAttacked(board, e1, b) && !DevIsSquareAttacked(board, d1, b))
                                DevAddMove(board, info, list, DevSetMove(e1, c1, 0, 0, 0, 0, 1));
                        }
                    }
                }
            }

            else
            {
                if (DevGetSq(fromSq) == bP)
                {
                    // pawn quiet move
                    if (!((fromSq - 16) & 0x88) && !DevGetSq(fromSq - 16))
                    {
                        if (rank_2 && !DevGetSq(fromSq - 16))
                        {
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq - 16, bN, 0, 0, 0, 0));
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq - 16, bB, 0, 0, 0, 0));
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq - 16, bR, 0, 0, 0, 0));
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq - 16, bQ, 0, 0, 0, 0));
                        }

                        else
                        {
                            DevAddMove(board, info, list, DevSetMove(fromSq, fromSq - 16, 0, 0, 0, 0, 0));

                            if (rank_7 && !DevGetSq(fromSq - 32))
                                DevAddMove(board, info, list, DevSetMove(fromSq, fromSq - 32, 0, 0, 1, 0, 0));
                        }
                    }

                    // pawn capture move
                    for (int i = 0; i < 4; ++i)
                    {
                        int dir = fromSq + dev_pawnAttacks[i];

                        // en passant move
                        if (dev_pawnAttacks[i] < 0 && !(dir & 0x88))
                        {
                            if (enPassant != noSq)
                            {
                                if (dir == enPassant)
                                    DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 0, 0, 1, 0));
                            }
                        }

                        if ((dev_pawnAttacks[i] < 0) && !(dir & 0x88) && isWhitePiece(dir))
                        {
                            if (rank_2)
                            {
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, bN, 1, 0, 0, 0));
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, bB, 1, 0, 0, 0));
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, bR, 1, 0, 0, 0));
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, bQ, 1, 0, 0, 0));
                            }

                            else
                            {
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 1, 0, 0, 0));
                            }
                        }
                    }
                }

                // castling
                if (DevGetSq(fromSq) == bK)
                {
                    if (castle & k)
                    {
                        if (!DevGetSq(f8) && !DevGetSq(g8))
                        {
                            if (!DevIsSquareAttacked(board, e8, w) && !DevIsSquareAttacked(board, f8, w))
                                DevAddMove(board, info, list, DevSetMove(e8, g8, 0, 0, 0, 0, 1));
                        }
                    }

                    if (castle & q)
                    {
                        if (!DevGetSq(d8) && !DevGetSq(c8) && !DevGetSq(b8))
                        {
                            if (!DevIsSquareAttacked(board, e8, w) && !DevIsSquareAttacked(board, d8, w))
                                DevAddMove(board, info, list, DevSetMove(e8, c8, 0, 0, 0, 0, 1));
                        }
                    }
                }
            }

            // knights
            if (side ? DevGetSq(fromSq) == bN : DevGetSq(fromSq) == wN)
            {
                for (int i = 0; i < 8; ++i)
                {
                    int dir = sq + dev_knightAttacks[i];
                    int delta = DevGetSq(dir);

                    if (!(dir & 0x88))
                    {
                        if (side ? (!delta || isWhitePiece(dir)) : (!delta || isBlackPiece(dir)))
                        {
                            if (!delta)
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 0, 0, 0, 0));
                            else
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 1, 0, 0, 0));
                        }
                    }
                }
            }

            // bishops and queens
            if (side ? (DevGetSq(fromSq) == bB) || (DevGetSq(fromSq) == bQ) : (DevGetSq(fromSq) == wB) || (DevGetSq(fromSq) == wQ))

            {
                for (int i = 0; i < 4; ++i)
                {
                    int dir = sq + dev_bishopAttacks[i];

                    while (!(dir & 0x88))
                    {
                        int delta = DevGetSq(dir);

                        // if hits own piece
                        if (side ? isBlackPiece(dir) : isWhitePiece(dir))
                            break;

                        // if hits opponent's piece
                        else if (side ? isWhitePiece(dir) : isBlackPiece(dir))
                        {
                            DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 1, 0, 0, 0));
                            break;
                        }

                        // on empty square
                        else if (!delta)
                        {
                            DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 0, 0, 0, 0));
                        }

                        dir += dev_bishopAttacks[i];
                    }
                }
            }

            // rooks and queens
            if (side ? (DevGetSq(fromSq) == bR) || (DevGetSq(fromSq) == bQ) : (DevGetSq(fromSq) == wR) || (DevGetSq(fromSq) == wQ))

            {
                for (int i = 0; i < 4; ++i)
                {
                    int dir = sq + dev_rookAttacks[i];

                    while (!(dir & 0x88))
                    {
                        int delta = DevGetSq(dir);

                        // if hits own piece
                        if (side ? isBlackPiece(dir) : isWhitePiece(dir))
                            break;

                        // if hits opponent's piece
                        else if (side ? isWhitePiece(dir) : isBlackPiece(dir))
                        {
                            DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 1, 0, 0, 0));
                            break;
                        }

                        // on empty square
                        else if (!delta)
                        {
                            DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 0, 0, 0, 0));
                        }

                        dir += dev_rookAttacks[i];
                    }
                }
            }

            // kings
            if (side ? DevGetSq(fromSq) == bK : DevGetSq(fromSq) == wK)
            {
                for (int i = 0; i < 8; ++i)
                {
                    int dir = sq + dev_kingAttacks[i];
                    int delta = DevGetSq(dir);

                    if (!(dir & 0x88))
                    {
                        if (side ? (!delta || isWhitePiece(dir)) : (!delta || isBlackPiece(dir)))
                        {
                            if (!delta)
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 0, 0, 0, 0));
                            else
                                DevAddMove(board, info, list, DevSetMove(fromSq, dir, 0, 1, 0, 0, 0));
                        }
                    }
                }
            }
        }
    }
}

// Function that makes a Move and returns a number based on whether or not the move is legal
// The function works by taking the move and checking whether or not its legal, and if it is
// then the function will change the board position
static inline int MakeMove(Chessboard *board, int move, int capFlag)
{
    // if capFlag make only captures else make all

    if (!capFlag)
    {
        ply++;

        Chessboard boardStored[1];
        boardStored[0] = board[0];

        int fromSq = GetMoveSource(move);
        int toSq = GetMoveTarget(move);

        // move piece
        GetSq(toSq) = GetSq(fromSq);
        GetSq(fromSq) = emSq;

        // promotions
        if (GetMovePromPiece(move))
        {
            GetSq(toSq) = GetMovePromPiece(move);
            GetSq(fromSq) = emSq;
        }

        // en passant flag
        if (GetMoveEnPassantFlag(move))
        {
            side ? (GetSq(enPassant + 16) = 0) : (GetSq(enPassant - 16) = 0);

            enPassant = noSq;
        }

        enPassant = noSq;

        // pawn start flag
        if (GetMovePawnStartFlag(move))
        {
            side ? (enPassant = toSq + 16) : (enPassant = toSq - 16);
        }

        // castling flag
        if (GetMoveCastleFlag(move))
        {
            switch (toSq)
            {
            case g1:
                GetSq(f1) = GetSq(h1);
                GetSq(h1) = emSq;
                break;

            case c1:
                GetSq(d1) = GetSq(a1);
                GetSq(a1) = emSq;
                break;

            case g8:
                GetSq(f8) = GetSq(h8);
                GetSq(h8) = emSq;
                break;

            case c8:
                GetSq(d8) = GetSq(a8);
                GetSq(a8) = emSq;
                break;
            }
        }

        // update castling permission
        castle &= castling[fromSq];
        castle &= castling[toSq];

        // update kingSq
        if (GetSq(GetMoveTarget(move)) == wK || GetSq(GetMoveTarget(move)) == bK)
            kingSq(side) = GetMoveTarget(move);

        // change side
        side ^= 1;

        // take back if king is in check
        if (InCheck(board, side ^ 1))
        {
            TakeBack(board, boardStored);
            return 0;
        }

        else
            return 1;
    }

    else
    {
        if (GetMoveCaptureFlag(move))
            MakeMove(board, move, allPos);
        else
            return 0;
    }

    return 0;
}

// Device version of function above
__device__ static inline int DevMakeMove(Chessboard *board, int move, int capFlag)
{
    // if capFlag make only captures else make all

    if (!capFlag)
    {
        ply++;

        Chessboard boardStored[1];
        boardStored[0] = board[0];

        int fromSq = DevGetMoveSource(move);
        int toSq = DevGetMoveTarget(move);

        // move piece
        DevGetSq(toSq) = DevGetSq(fromSq);
        DevGetSq(fromSq) = emSq;

        // promotions
        if (DevGetMovePromPiece(move))
        {
            DevGetSq(toSq) = DevGetMovePromPiece(move);
            DevGetSq(fromSq) = emSq;
        }

        // en passant flag
        if (DevGetMoveEnPassantFlag(move))
        {
            side ? (DevGetSq(enPassant + 16) = 0) : (DevGetSq(enPassant - 16) = 0);

            enPassant = noSq;
        }

        enPassant = noSq;

        // pawn start flag
        if (DevGetMovePawnStartFlag(move))
        {
            side ? (enPassant = toSq + 16) : (enPassant = toSq - 16);
        }

        // castling flag
        if (DevGetMoveCastleFlag(move))
        {
            switch (toSq)
            {
            case g1:
                DevGetSq(f1) = DevGetSq(h1);
                DevGetSq(h1) = emSq;
                break;

            case c1:
                DevGetSq(d1) = DevGetSq(a1);
                DevGetSq(a1) = emSq;
                break;

            case g8:
                DevGetSq(f8) = DevGetSq(h8);
                DevGetSq(h8) = emSq;
                break;

            case c8:
                DevGetSq(d8) = DevGetSq(a8);
                DevGetSq(a8) = emSq;
                break;
            }
        }

        // update castling permission
        castle &= dev_castling[fromSq];
        castle &= dev_castling[toSq];

        // update kingSq
        if (DevGetSq(DevGetMoveTarget(move)) == wK || DevGetSq(DevGetMoveTarget(move)) == bK)
            DevkingSq(side) = DevGetMoveTarget(move);

        // change side
        side ^= 1;

        // take back if king is in check
        if (DevInCheck(board, side ^ 1))
        {
            DevTakeBack(board, boardStored);
            return 0;
        }

        else
            return 1;
    }

    else
    {
        if (DevGetMoveCaptureFlag(move))
            DevMakeMove(board, move, allPos);
        else
            return 0;
    }

    return 0;
}

// Evaluates the position
// The function loops through all the pieces on the board and
// adds the material weight of every piece to a sum
// the material weight is defined by the tables at the top
// which are the stockfish analysis tables.
static inline int EvaluatePosition(Chessboard *board)
{
    int score = 0;

    for (int sq = 0; sq < 128; ++sq)
    {
        if (!(sq & 0x88) && GetSq(sq))
        {
            // evaluate material
            score += materialWeight[GetSq(sq)];

            // evaluate piece placement
            switch (GetSq(sq))
            {
            case wP:
                score += Pawns[sq];
                break;

            case wN:
                score += Knights[sq];
                break;

            case wB:
                score += Bishops[sq];
                break;

            case wR:
                score += Rooks[sq];
                break;

            case wK:
                score += Kings[sq];
                break;

            case bP:
                score -= Pawns[MirrorSq(sq)];
                break;

            case bN:
                score -= Knights[MirrorSq(sq)];
                break;

            case bB:
                score -= Bishops[MirrorSq(sq)];
                break;

            case bR:
                score -= Rooks[MirrorSq(sq)];
                break;

            case bK:
                score -= Kings[MirrorSq(sq)];
                break;
            }
        }
    }

    if (!side)
        return score;

    else
        return -score;
}

// Device repeat function
__device__ static inline int DevEvaluatePosition(Chessboard *board)
{
    int score = 0;

    for (int sq = 0; sq < 128; ++sq)
    {
        if (!(sq & 0x88) && GetSq(sq))
        {
            // evaluate material
            score += dev_materialWeight[GetSq(sq)];

            // evaluate piece placement
            switch (GetSq(sq))
            {
            case wP:
                score += dev_Pawns[sq];
                break;

            case wN:
                score += dev_Knights[sq];
                break;

            case wB:
                score += dev_Bishops[sq];
                break;

            case wR:
                score += dev_Rooks[sq];
                break;

            case wK:
                score += dev_Kings[sq];
                break;

            case bP:
                score -= dev_Pawns[DevMirrorSq(sq)];
                break;

            case bN:
                score -= dev_Knights[DevMirrorSq(sq)];
                break;

            case bB:
                score -= dev_Bishops[DevMirrorSq(sq)];
                break;

            case bR:
                score -= dev_Rooks[DevMirrorSq(sq)];
                break;

            case bK:
                score -= dev_Kings[DevMirrorSq(sq)];
                break;
            }
        }
    }

    if (!side)
        return score;

    else
        return -score;
}

// Sets all of the Search variables to 0
void InitSearch(Search *info)
{
    info->nodes = 0;
    info->fhf = 0;
    info->fh = 0;
    info->bestScore = 0;
}

// NegaMaxSearch that runs entirely on the CPU, for comparison purposes, explained in doc
static int RegNegaMaxSearch(Chessboard *board, Search *info, int depth)
{
    int bestMove = 0;
    int alpha = -50000;
    int oldAlpha = alpha;
    int score = -50000;
    int legalMoves = 0;

    info->nodes++;

    if (depth == 0)
        return EvaluatePosition(board);

    Movelist list[1];
    GenerateMoves(board, info, list);

    // loops through all possible moves, recurssively calls function
    for (int moveNum = 0; moveNum < list->moveCount; ++moveNum)
    {
        Chessboard boardStored[1];
        boardStored[0] = board[0];

        SortMoves;

        if (!MakeMove(board, list->moves[moveNum].move, allPos))
            continue;

        legalMoves++;
        score = -RegNegaMaxSearch(board, info, depth - 1);
        TakeBack(board, boardStored);

        if (score > alpha)
        {
            alpha = score;
            bestMove = list->moves[moveNum].move;
        }
    }

    if (!legalMoves)
    {
        if (InCheck(board, side))
            return -49000 + ply; // on checkmate

        else
            return 0; // on stalemate
    }

    if (alpha != oldAlpha)
    {
        info->bestMove = bestMove;
    }

    return alpha;
}

// NegaMaxSearch used when CPU depth is above 2, function is not in use anywhere as CPUdepth > 2 is not allowed
static int EvalNegaMaxSearch(Chessboard *board, Search *info, Search *searchP, int depth, int &count)
{
    int bestMove = 0;
    int alpha = -50000;
    int oldAlpha = alpha;
    int score = -50000;
    int legalMoves = 0;

    info->nodes++;

    if (depth == 0)
    {
        count++;
        return searchP[count - 1].bestScore;
    }

    Movelist list[1];
    GenerateMoves(board, info, list);

    // the good loop
    for (int moveNum = 0; moveNum < list->moveCount; ++moveNum)
    {
        Chessboard boardStored[1];
        boardStored[0] = board[0];

        SortMoves;

        if (!MakeMove(board, list->moves[moveNum].move, allPos))
            continue;

        legalMoves++;
        score = -EvalNegaMaxSearch(board, info, searchP, depth - 1, count);
        TakeBack(board, boardStored);

        if (score > alpha)
        {
            alpha = score;
            bestMove = list->moves[moveNum].move;
        }
    }

    if (!legalMoves)
    {
        if (InCheck(board, side))
            return -49000 + ply; // on checkmate

        else
            return 0; // on stalemate
    }

    if (alpha != oldAlpha)
    {
        info->bestMove = bestMove;
    }

    return alpha;
}

// Same as CreateNegaMaxSearch except dcount is no longer here and since we aren't at the top depth validMoves are not being added
static void ContinueNegaMaxSearch(Chessboard *board, Search *info, int depth, int &count)
{

    info->nodes++;

    if (depth == 0)
    {

        Search push;
        InitSearch(&push);

        boards[count] = board[0];
        searches.push_back(push);

        count++;

        return;
    }

    Movelist list[1];

    GenerateMoves(board, info, list);

    // the good loop
    for (int moveNum = 0; moveNum < list->moveCount; ++moveNum)
    {
        Chessboard boardStored[1];
        boardStored[0] = board[0];

        if (!MakeMove(board, list->moves[moveNum].move, allPos))
            continue;

        ContinueNegaMaxSearch(board, info, depth - 1, count);
        // store the count in a vector

        TakeBack(board, boardStored);
    }

    return;
}

//NegaMaxSearch style searchwith the adding of the board positions and info to the vectors.
static void CreateNegaMaxSearch(Chessboard *board, Search *info, Move *valid, int *moveCounterPtr, int depth, int &count, int &dcount)
{

    info->nodes++;

    if (depth == 0)
    {

        Search push;
        InitSearch(&push);

        boards[count] = board[0];
        searches.insert(searches.begin(), push);

        count++;
        return;
    }

    Movelist list[1];

    GenerateMoves(board, info, list);

    // Loops through all possible moves at this level and recursively calls itself with 1 less depth
    for (int moveNum = 0; moveNum < list->moveCount; ++moveNum)
    {
        Chessboard boardStored[1];
        boardStored[0] = board[0];

        if (!MakeMove(board, list->moves[moveNum].move, allPos))
            continue;

        // Adds valid moves to array for analysis after GPU search
        valid[dcount].move = list->moves[moveNum].move;

        ContinueNegaMaxSearch(board, info, depth - 1, count);

        // Counter used for searching when CPUdepth = 2 to do a MiniMax search
        moveCounterPtr[dcount] = count;
        //moveCounter->push_back(count[0]);
        dcount++;

        TakeBack(board, boardStored);
    }
}

// NegaMaxSearch that the GPU should call, Same as RegNegaMax
__device__ static int SplitNegaMaxSearch(Chessboard *board, Search *info, int depth)
{

    int bestMove = 0;
    int alpha = -50000;
    int oldAlpha = alpha;
    int score = -50000;
    int legalMoves = 0;

    info->nodes++;

    if (depth == 0)
    {
        //printf("ep ");
        return DevEvaluatePosition(board);
    }

    Movelist list[1];

    DevGenerateMoves(board, info, list);

    // the good loop
    for (int moveNum = 0; moveNum < list->moveCount; ++moveNum)
    {
        Chessboard boardStored[1];
        boardStored[0] = board[0];

        SortMoves;

        if (!DevMakeMove(board, list->moves[moveNum].move, allPos))
            continue;

        legalMoves++;
        score = -SplitNegaMaxSearch(board, info, depth - 1);

        // this feels really inefficient as we could just DevMakeMove the same move backward
        DevTakeBack(board, boardStored);

        if (score > alpha)
        {
            alpha = score;
            bestMove = list->moves[moveNum].move;
        }
    }

    if (!legalMoves)
    {
        if (DevInCheck(board, side))
            return -49000 + ply; // on checkmate

        else
            return 0; // on stalemate
    }

    if (alpha != oldAlpha)
    {
        info->bestMove = bestMove;
        info->bestScore = alpha;
    }

    return alpha;
}

// Kernel function called that starts searching assuming the thread number is in our needed array of searches
__global__ static void kernelSearch(Chessboard *dev_board, Search *dev_searches, int *dev_totalThreadCount, int *gpu_depth)
{

    int location = 256 * blockIdx.x + threadIdx.x;
    int depth = gpu_depth[0];
    //printf("ker: %i\n", location);

    // checks if this thread needs to calculate
    if (location < dev_totalThreadCount[0])
    {

        SplitNegaMaxSearch(&dev_board[location], &dev_searches[location], depth); 
        //printf("done");
    }
}

//The Search algorithm that calls and manages everything
static int GPUNegaMaxSearch(Chessboard *board, Search *info, int cpu_depth, int gpu_depth)
{

    // Used for when cpuDepth is 2
    int moveCounter[146];
    boards.reserve(pow(100, 3));
    searches.reserve(pow(100, 3)); 

    // initalize the variables to 0
    int totalThreadCount = 0;
    int dcount = 0;
    Movelist saveList[1];
    saveList->moveCount = 0;
    GenerateMoves(board, info, saveList);

    Move validMoves[256];
    Move *validMovePtr = validMoves;

    int *moveCounterPtr = moveCounter;

    // Creates and saves the boards we need to run on the GPU
    CreateNegaMaxSearch(board, info, validMovePtr, moveCounterPtr, cpu_depth, totalThreadCount, dcount);

    // Calculate the block count and total threads per block
    double result = (double)totalThreadCount / (double)256;
    int blockCount = (int)(ceil(result));
    //printf("blockCount: %i", blockCount);
    int threadsPerBlock = 256;
    if (totalThreadCount < threadsPerBlock)
        threadsPerBlock = totalThreadCount;

    //printf("TTC: %i\n", totalThreadCount);

    // Copy arrys over to the device
    Chessboard *boardP;
    Search *searchP;

    boardP = (Chessboard *)malloc(totalThreadCount * sizeof(Chessboard));
    searchP = (Search *)malloc(totalThreadCount * sizeof(Search));

    Chessboard *dev_boards;
    Search *dev_searches;
    int *dev_totalThreadCount;
    int *dev_gpuDepth;

    //allocate space for boards and searchers
    hipMalloc((void **)&dev_boards, totalThreadCount * sizeof(Chessboard));
    hipMalloc((void **)&dev_searches, totalThreadCount * sizeof(Search));
    hipMalloc((void **)&dev_totalThreadCount, sizeof(int));
    hipMalloc((void **)&dev_gpuDepth, sizeof(int));

    // copy vectors to arrays
    for (int i = 0; i < totalThreadCount; i++)
    {
        boardP[i] = boards[i];
        searchP[i] = searches[i];
    }

    // copy arrays to the GPU
    hipMemcpy(dev_boards, boardP, totalThreadCount * sizeof(Chessboard), hipMemcpyHostToDevice);
    hipMemcpy(dev_searches, searchP, totalThreadCount * sizeof(Search), hipMemcpyHostToDevice);
    hipMemcpy(dev_totalThreadCount, &totalThreadCount, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_gpuDepth, &gpu_depth, sizeof(int), hipMemcpyHostToDevice);

    // CUDA cannot calculate recursive stack size, so we have to set stack size manually
    size_t limit = 2616*(gpu_depth+1); //space taken up by going one deeper, plus the first depth
    hipDeviceSetLimit(hipLimitStackSize, limit);

    // for testing
    //threadsPerBlock = 256;

    // create variables and then run our search
    dim3 grid(blockCount, 1, 1);

    kernelSearch<<<grid, threadsPerBlock>>>(dev_boards, dev_searches, dev_totalThreadCount, dev_gpuDepth);
    hipDeviceSynchronize();

    // copy the data back to the arrays

    hipMemcpy(boardP, dev_boards, totalThreadCount * sizeof(Chessboard), hipMemcpyDeviceToHost);
    hipMemcpy(searchP, dev_searches, totalThreadCount * sizeof(Search), hipMemcpyDeviceToHost);

    hipFree(dev_boards);
    hipFree(dev_searches);

    int retScore = -50000;
    int bestIndex = 0;

    if (cpu_depth == 1)
    {
        // Simple Search
        int reverse = -1;

        for (int i = 0; i < totalThreadCount; i++)
        {
            // printf("Index: %i Move: ", i);
            // PrintMove(searchP[i].bestMove);
            // printf(" Score: %i \n", searchP[i].bestScore);
            if (reverse * searchP[i].bestScore > retScore)
            {
                retScore = reverse * searchP[i].bestScore;
                bestIndex = i;

                //printf("index: %i ", bestIndex);
                //PrintMove(searchP[i].bestMove);
                //printf("\n");
            }
        }

        info->bestMove = validMoves[bestIndex].move;
    }
    else if (cpu_depth == 2)
    {
        // MiniMax adjacent search to determine our best move
        int lastIndex = 0;
        int maxVal = -90000;
        int minVal = 90000;
        for (int i = 0; i < dcount; i++)
        {
            minVal = 90000;
            for (int j = lastIndex; j < moveCounter[i]; j++)
            {
                if (searchP[j].bestScore < minVal)
                {
                    minVal = searchP[j].bestScore;
                }
            }
            if (minVal > maxVal)
            {
                maxVal = minVal;
                bestIndex = i;
            }
            lastIndex = moveCounter[i];
        }
        retScore = maxVal;
        info->bestMove = validMoves[bestIndex].move;
    }
    else
    {
        // This is logically sound as the evalutation function just becomes the array index of that position
        // but will not launch as the number of threads will be too high
        int count = 0;
        retScore = EvalNegaMaxSearch(board, info, searchP, cpu_depth, count);
    }

    // printf("index: %i \n", bestIndex);

    free(boardP);
    free(searchP);

    return retScore;
}

// Function that runs our search and times our functions
static inline void SearchPosition(Chessboard *board, Search *info, int cpuDepth, int gpuDepth)
{

    clock_t start, end;
    start = clock();
    int score;

    if (gpuDepth > 0)
    {
        score = GPUNegaMaxSearch(board, info, cpuDepth, gpuDepth);
    }
    else
    {
        score = RegNegaMaxSearch(board, info, cpuDepth);
    }

    end = clock();
    double time = (double)(end - start) / CLOCKS_PER_SEC;

    if (score == 49000)
        return;

    printf("info score cp %d depth %d depth %d\n", score, cpuDepth, gpuDepth);
    printf("Time taken to depth %f seconds\n", time);

    FILE *out_file = fopen("out_file.txt", "a"); // write only
    fprintf(out_file, "%f \n", time);
    fclose(out_file);

    printf("bestmove ");
    PrintMove(info->bestMove);
    printf("\n");

    //printf("Move ordering: %.2f\n",(info->fhf/info->fh));
}


// Function that converts a FEN into a board
void ParseFen(Chessboard *board, char *fen)
{
    ResetBoard(board);

    RankLoop{
        FileLoop{
            int sq = fr2sq(file, rank);

    // parse position
    if (IsOnBoard(sq))
    {
        if (isPieceChar(*fen))
        {
            if (*fen == 'K')
                kingSq(w) = sq;

            else if (*fen == 'k')
                kingSq(b) = sq;

            //printf( "%i", *fen);
            //printf("Here");

            switch (*fen)
            {

            case 114:
                SetSq(sq, 12);
                break; // wR
            case 110:
                SetSq(sq, 10);
                break; // wN
            case 98:
                SetSq(sq, 11);
                break; // wB
            case 113:
                SetSq(sq, 13);
                break; // wQ
            case 112:
                SetSq(sq, 9);
                break; // wP

            case 82:
                SetSq(sq, 4);
                break; //bR
            case 78:
                SetSq(sq, 2);
                break; //bN
            case 66:
                SetSq(sq, 3);
                break; //bB
            case 81:
                SetSq(sq, 5);
                break; //bQ
            case 80:
                SetSq(sq, 1);
                break; //bP
            }

            *fen++;
        }

        if (isDigit(*fen))
        {
            int count = *fen - '0';

            if (!GetSq(sq))
                file++;

            file -= count;
            *fen++;
        }

        if (*fen == '/')
        {
            *fen++;
            file--;
        }
    }
}
}

*fen++;

// parse stats
side = (*fen == 'w') ? w : b;
fen += 2;

while (*fen != ' ')
{
    switch (*fen)
    {
    case 'K':
        castle |= K;
        break;
    case 'Q':
        castle |= Q;
        break;
    case 'k':
        castle |= k;
        break;
    case 'q':
        castle |= q;
        break;

    case '-':
        castle = 0;
    }
    fen++;
}

fen++;

if (*fen != '-')
{
    int file = fen[0] - 'a';
    int rank = fen[1] - '0';
    enPassant = parse2sq(file, rank);
}
}

// Parses a move
int ParseMove(Chessboard *board, Search *info, char *moveStr)
{
    Movelist list[1];
    GenerateMoves(board, info, list);

    int parseFrom = (moveStr[0] - 'a') + (moveStr[1] - '0' - 1) * 16;
    int parseTo = (moveStr[2] - 'a') + (moveStr[3] - '0' - 1) * 16;
    int promPiece = 0;

    int move;

    for (int moveNum = 0; moveNum < list->moveCount; ++moveNum)
    {
        move = list->moves[moveNum].move;

        if (GetMoveSource(move) == parseFrom && GetMoveTarget(move) == parseTo)
        {
            promPiece = GetMovePromPiece(move);

            if (promPiece)
            {
                if ((promPiece == wN || promPiece == bN) && moveStr[4] == 'n')
                    return move;

                else if ((promPiece == wB || promPiece == bB) && moveStr[4] == 'b')
                    return move;

                else if ((promPiece == wR || promPiece == bR) && moveStr[4] == 'r')
                    return move;

                else if ((promPiece == wQ || promPiece == bQ) && moveStr[4] == 'q')
                    return move;

                continue;
            }

            return move;
        }
    }

    return 0;
}

// Performance testing code when "test" is called, Search Position but loops through file for input
static inline void TestSearchPosition(Chessboard *board, Search *info, int cpuDepth, int gpuDepth)
{

    char const *const fileName = "test.txt"; 
    FILE *file = fopen(fileName, "r");
    char line[256];

    while (fgets(line, sizeof(line), file))
    {
        ParseFen(board, line);
        InitSearch(info);
        PrintBoard(board);

        clock_t start, end;
        start = clock();

        int score;
        if (gpuDepth > 0)
        {
            score = GPUNegaMaxSearch(board, info, cpuDepth, gpuDepth);
        }
        else
        {
            score = RegNegaMaxSearch(board, info, cpuDepth);
        }

        end = clock();
        double time = (double)(end - start) / CLOCKS_PER_SEC;

        if (score == 49000)
            return;

        printf("info score cp %d depth %d depth %d\n", score, cpuDepth, gpuDepth);
        printf("Time taken to depth %f seconds\n", time);

        FILE *out_file = fopen("out_file.txt", "a"); // write only
        fprintf(out_file, "%f \n", time);
        fclose(out_file);

        printf("bestmove ");
        PrintMove(info->bestMove);
        printf("\n");
    }
}

// Hammerhead main
int main(int argc, char *argv[])
{
    // Init everything
    Chessboard board[1];
    Search info[1];
    InitSearch(info);

    char inFen[87];

    int cpuDepth = 2;
    int gpuDepth = 3;

    // If user specifies depth, set depths
    if (argc == 3)
    {
        cpuDepth = atoi(argv[1]);
        gpuDepth = atoi(argv[2]);
    }

    // Larger cpuDepth is not yet implemented so this is a failsafe
    if (cpuDepth > 2){
        cpuDepth = 2;
    }

    // Loop through accepting and analyzing FENs
    while (1)
    {

        printf("Please enter a fen to analyze: \n");

        // take input fen
        fgets(inFen, 87, stdin);

        if (std::strstr(inFen, "quit") || std::strstr(inFen, "done") || std::strstr(inFen, "over"))
        {
            return 0;
        }

        if (std::strstr(inFen, "test"))
        {
            TestSearchPosition(board, info, cpuDepth, gpuDepth);
            return 0;
        }

        ParseFen(board, inFen);

        PrintBoard(board);

        SearchPosition(board, info, cpuDepth, gpuDepth);

    }

    return 0;
}
